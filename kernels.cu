#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void gpu_Heat(float *h, float *g, int N, float *dev_residual) {
    // In this case, the stride is N

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Assert if current thread belongs to first row or column of input matrix
    bool is_first_row = threadIdx.y == 0 && blockIdx.y == 0;
    bool is_first_col = threadIdx.x == 0 && blockIdx.x == 0;

    if (i < N - 1 && j < N - 1 && !is_first_row && !is_first_col) {
        int pos = i * N + j;
        g[pos] = 0.25 * (h[pos - 1] + h[pos - N] + h[pos + 1] + h[pos + N]);
        float diff = g[pos] - h[pos];
        atomicAdd(dev_residual, diff * diff);
    }
}
